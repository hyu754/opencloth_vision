#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include <iostream>
#include "AFEM_cuda.cuh"
#include "Utilities.cuh"

#define IDX2C(i,j,ld) (((j)*(ld))+( i )) 
void printMatrix(int m, int n, const double*A, int lda, const char* name) { for (int row = 0; row < m; row++){ for (int col = 0; col < n; col++){ double Areg = A[row + col*lda]; printf("%s(%d,%d) = %f\n", name, row + 1, col + 1, Areg); } } }

void cuda_tools::initialize_cholesky_variables(int numNodes, int numElem, int dim){
	Nrows = numNodes*dim;                        // --- Number of rows
	Ncols = numNodes*dim;                        // --- Number of columns
	N = Nrows;
	cusparseSafeCall(hipsparseCreate(&handle));

	//h_A_dense = (float*)malloc(Nrows*Ncols*sizeof(*h_A_dense));
	cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	cusparseSafeCall(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
	cusparseSafeCall(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));
	nnz = 0;                                // --- Number of nonzero elements in dense matrix
	lda = Nrows;                      // --- Leading dimension of dense matrix
	gpuErrchk(hipMalloc(&d_nnzPerVector, Nrows * sizeof(*d_nnzPerVector)));
	h_nnzPerVector = (int *)malloc(Nrows * sizeof(*h_nnzPerVector));


	//device side dense matrix
	gpuErrchk(hipMalloc(&d_A_RowIndices, (Nrows + 1) * sizeof(*d_A_RowIndices)));



	//hipMemcpy(&numNodes,dev_numNodes , sizeof(dev_numNodes), hipMemcpyDeviceToHost);


	cusparseSafeCall(hipsparseCreateMatDescr(&descr_L));
	cusparseSafeCall(hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ONE));
	cusparseSafeCall(hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL));
	cusparseSafeCall(hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER));
	cusparseSafeCall(hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_NON_UNIT));

	//emeory in cholesky
	cusparseSafeCall(hipsparseCreateCsric02Info(&info_A));
	cusparseSafeCall(hipsparseCreateCsrsv2Info(&info_L));
	cusparseSafeCall(hipsparseCreateCsrsv2Info(&info_Lt));
}

void cuda_tools::cholesky()
{
	
#if 0
	std::ofstream writenodes("global_K.txt");

	for (int j = 0; j < N; j++){
		for (int i = 0; i < N; i++){
			writenodes << h_A_dense[IDX2C(j, i, N)] << " ";
		}
		writenodes << std::endl;
	}

	writenodes.close();
#endif // 0


	// --- Create device array and copy host array to it
	/*for (int j = 0; j < 20; j++){
	for (int i = 0; i < 20; i++){
	std::cout << h_A_dense[IDX2C(j, i, N)] << std::endl;
	}
	std::cout<<std::endl;
	}*/



	// --- Descriptor for sparse matrix A





	// --- Device side number of nonzero elements per row

	cusparseSafeCall(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, Nrows, Ncols, descrA, LHS, lda, d_nnzPerVector, &nnz));
	// --- Host side number of nonzero elements per row


	gpuErrchk(hipMemcpy(h_nnzPerVector, d_nnzPerVector, Nrows * sizeof(*h_nnzPerVector), hipMemcpyDeviceToHost));

	/*printf("Number of nonzero elements in dense matrix = %i\n\n", nnz);
	for (int i = 0; i < 10; ++i) printf("Number of nonzero elements in row %i = %i \n", i, h_nnzPerVector[i]);
	printf("\n");*/

	// --- Device side dense matrix
	gpuErrchk(hipMalloc(&d_A, nnz * sizeof(*d_A)));
	gpuErrchk(hipMalloc(&d_A_ColIndices, nnz * sizeof(*d_A_ColIndices)));

	
	cusparseSafeCall(hipsparseSdense2csr(handle, Nrows, Ncols, descrA, LHS, lda, d_nnzPerVector, d_A, d_A_RowIndices, d_A_ColIndices));
	// --- Host side dense matrix

	float *h_A = (float *)malloc(nnz * sizeof(*h_A));
	int *h_A_RowIndices = (int *)malloc((Nrows + 1) * sizeof(*h_A_RowIndices));
	int *h_A_ColIndices = (int *)malloc(nnz * sizeof(*h_A_ColIndices));
	gpuErrchk(hipMemcpy(h_A, d_A, nnz*sizeof(*h_A), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_A_RowIndices, d_A_RowIndices, (Nrows + 1) * sizeof(*h_A_RowIndices), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_A_ColIndices, d_A_ColIndices, nnz * sizeof(*h_A_ColIndices), hipMemcpyDeviceToHost));



	
//	std::cout << nnz << std::endl;

	/*printf("\nOriginal matrix in CSR format\n\n");
	for (int i = 0; i < 10; ++i) printf("A[%i] = %.0f ", i, h_A[i]); printf("\n");

	printf("\n");
	for (int i = 0; i < (10 + 1); ++i) printf("h_A_RowIndices[%i] = %i \n", i, h_A_RowIndices[i]); printf("\n");

	for (int i = 0; i < 10; ++i) printf("h_A_ColIndices[%i] = %i \n", i, h_A_ColIndices[i]);
	*/
	// --- Allocating and defining dense host and device data vectors

	//float *h_x = (float *)malloc(Nrows * sizeof(float));
	///*h_x[0] = 100.0;  h_x[1] = 200.0; h_x[2] = 400.0; h_x[3] = 500.0;*/
	//for (int i = 0; i < N; i++){
	//	h_x[i] = 0.00001;
	//}


	float *d_x;        gpuErrchk(hipMalloc(&d_x, Nrows * sizeof(float)));
	gpuErrchk(hipMemcpy(d_x, RHS, Nrows * sizeof(float), hipMemcpyHostToDevice));



	/******************************************/
	/* STEP 1: CREATE DESCRIPTORS FOR L AND U */
	/******************************************/




	/********************************************************************************************************/
	/* STEP 2: QUERY HOW MUCH MEMORY USED IN CHOLESKY FACTORIZATION AND THE TWO FOLLOWING SYSTEM INVERSIONS */
	/********************************************************************************************************/


	int pBufferSize_M, pBufferSize_L, pBufferSize_Lt;
	cusparseSafeCall(hipsparseScsric02_bufferSize(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, &pBufferSize_M));
	cusparseSafeCall(hipsparseScsrsv2_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, &pBufferSize_L));
	cusparseSafeCall(hipsparseScsrsv2_bufferSize(handle, HIPSPARSE_OPERATION_TRANSPOSE, N, nnz, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_Lt, &pBufferSize_Lt));

	int pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_Lt));
	void *pBuffer = 0;  gpuErrchk(hipMalloc((void**)&pBuffer, pBufferSize));


	/******************************************************************************************************/
	/* STEP 3: ANALYZE THE THREE PROBLEMS: CHOLESKY FACTORIZATION AND THE TWO FOLLOWING SYSTEM INVERSIONS */
	/******************************************************************************************************/
	int structural_zero;

	cusparseSafeCall(hipsparseScsric02_analysis(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer));

	hipsparseStatus_t status = hipsparseXcsric02_zeroPivot(handle, info_A, &structural_zero);
	if (HIPSPARSE_STATUS_ZERO_PIVOT == status){ printf("A(%d,%d) is missing\n", structural_zero, structural_zero); }

	cusparseSafeCall(hipsparseScsrsv2_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer));
	cusparseSafeCall(hipsparseScsrsv2_analysis(handle, HIPSPARSE_OPERATION_TRANSPOSE, N, nnz, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_Lt, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer));

	/*************************************/
	/* STEP 4: FACTORIZATION: A = L * L' */
	/*************************************/
	int numerical_zero;

	cusparseSafeCall(hipsparseScsric02(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer));
	status = hipsparseXcsric02_zeroPivot(handle, info_A, &numerical_zero);
	/*if (HIPSPARSE_STATUS_ZERO_PIVOT == status){ printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero); }
	*/

	gpuErrchk(hipMemcpy(h_A, d_A, nnz * sizeof(float), hipMemcpyDeviceToHost));
	/*printf("\nNon-zero elements in Cholesky matrix\n\n");
	for (int k = 0; k<10; k++) printf("%f\n", h_A[k]);*/


	cusparseSafeCall(hipsparseScsr2dense(handle, Nrows, Ncols, descrA, d_A, d_A_RowIndices, d_A_ColIndices, LHS, Nrows));


	/*printf("\nCholesky matrix\n\n");
	for (int i = 0; i < 10; i++) {
	std::cout << "[ ";
	for (int j = 0; j < 10; j++)
	std::cout << h_A_dense[i * Ncols + j] << " ";
	std::cout << "]\n";
	}*/

	/*********************/
	/* STEP 5: L * z = x */
	/*********************/
	// --- Allocating the intermediate result vector
	float *d_z;        gpuErrchk(hipMalloc(&d_z, N * sizeof(float)));

	const float alpha = 1.;
	cusparseSafeCall(hipsparseScsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, &alpha, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, d_x, d_z, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer));

	/**********************/
	/* STEP 5: L' * y = z */
	/**********************/
	// --- Allocating the host and device side result vector
	float *h_y = (float *)malloc(Ncols * sizeof(float));
	float *d_y;        gpuErrchk(hipMalloc(&d_y, Ncols * sizeof(float)));

	cusparseSafeCall(hipsparseScsrsv2_solve(handle, HIPSPARSE_OPERATION_TRANSPOSE, N, nnz, &alpha, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_Lt, d_z, d_y, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer));

	//hipMemcpy(h_x, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
	/*for (int k = 0; k<20; k++) printf("dx[%i] = %f\n", k, h_x[k]);
	for (int k = 0; k<20; k++) printf("xs[%i] = %f\n", k, x[k]);*/
	
	hipMemcpy(h_y, LHS, Ncols * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < 4; i++){
		for (int j = 0; j < 4; j++){
			std::cout << h_y[IDX2C(j,i,Ncols)] << " ";
		}
		std::cout << std::endl;
	}
	
	std::cout << std::endl;
	
	update_geometry(d_y);


	hipFree(d_A);
	hipFree(d_A_ColIndices);
	hipFree(pBuffer);

	hipFree(d_z);
	hipFree(d_y);
	free(h_y);
	//free(h_x);


#if 0

	free(h_A);
	free(h_A_RowIndices);
	free(h_A_ColIndices);
	free(h_x);
	free(h_y);
	hipFree(d_x);
	hipFree(pBuffer);
	hipFree(d_z);
	hipFree(d_y);


	for (int i = 0; i < numNodes; i++) {
		x[i] = x[i] + h_x[i * dim];
		y[i] = y[i] + h_x[i * dim + 1];
		if (dim == 3){
			z[i] = z[i] + h_x[i * dim + 2];
		}

	}

	
	duration_K = (std::clock() - start_K) / (double)CLOCKS_PER_SEC;
	//std::cout << " change status : " << changeNode << std::endl;

	//std::cout << "FPS time: " <<1/duration_K << std::endl;

	//std::cout << "Duration: " << duration_K << std::endl;
	return 0;
#endif // 0

}
