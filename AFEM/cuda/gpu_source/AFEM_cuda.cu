#include "hip/hip_runtime.h"


#include <stdio.h>


#include <math.h>

#include <iostream>
#include "AFEM_cuda.cuh"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define nodesinelemX(node,el,nodesPerElem) (node + nodesPerElem*el) //the first entry is the element # the second entry would be the element number and the last one is the number of nodes/element
#define threeD21D(row_d,col_d,el_d,width_d,depth_d) (row_d+width_d*(col_d+depth_d*el_d)) //
#define nodesDisplacementX(dof,node,dimension) (dof + node*dimension)
#define IDX2C(i,j,ld) (((j)*(ld))+( i )) 


__global__ void gpu_print_vec(AFEM::element *in_vec){
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	printf("%d", in_vec[x].nodes_in_elem[0]);
}
__device__ void print_kernel() {
	printf("Hello from block");
}
__device__ void find_Jacobian_and_localK(AFEM::element *in_element){
	
	float x14 = in_element->position_info[0].x - in_element->position_info[3].x;
	float x24 = in_element->position_info[1].x - in_element->position_info[3].x;
	float x34 = in_element->position_info[2].x - in_element->position_info[3].x;
	float y14 = in_element->position_info[0].y - in_element->position_info[3].y;
	float y24 = in_element->position_info[1].y - in_element->position_info[3].y;
	float y34 = in_element->position_info[2].y - in_element->position_info[3].y;
	float z14 = in_element->position_info[0].z - in_element->position_info[3].z;
	float z24 = in_element->position_info[1].z - in_element->position_info[3].z;
	float z34 = in_element->position_info[2].z - in_element->position_info[3].z;

	//std::cout << x14*(y24*z34 - y34*z24) - y14*(x24*z34 - z24 * 34) + z14*(x24*y34 - y24*x34) << std::endl;
	float det_J = (x14*(y24*z34 - y34*z24) - y14*(x24*z34 - z24 * x34) + z14*(x24*y34 - y24*x34));
	float J_bar11 = (y24*z34 - z24*y34) / det_J;
	float J_bar12 = (z14*y34 - y14*z34) / det_J;
	float J_bar13 = (y14*z24 - z14*y24) / det_J;
	float J_bar21 = (z24*x34 - x24*z34) / det_J;
	float J_bar22 = (x14*z34 - z14*x34) / det_J;
	float J_bar23 = (z14*x24 - x14*z24) / det_J;
	float J_bar31 = (x24*y34 - y24*x34) / det_J;
	float J_bar32 = (y14*x34 - x14*y34) / det_J;
	float J_bar33 = (x14*y24 - y14*x24) / det_J;

	float J_star1 = -(J_bar11 + J_bar12 + J_bar13);
	float J_star2 = -(J_bar21 + J_bar22 + J_bar23);
	float J_star3 = -(J_bar31 + J_bar32 + J_bar33);

	in_element->Jacobian = det_J;

	

	float E = 100000.0;
	float nu = 0.49;
	

	in_element->local_K[  0] = 0.166666666666667*E*J_bar11*J_bar11 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	//in_element->local_K[0] = det_J;
	in_element->local_K[  1] = 0.166666666666667*E*J_bar11*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  2] = 0.166666666666667*E*J_bar11*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  3] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  4] = 0.166666666666667*E*J_bar11*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  5] = 0.166666666666667*E*J_bar11*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  6] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  7] = 0.166666666666667*E*J_bar11*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  8] = 0.166666666666667*E*J_bar11*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  9] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  10] = 0.166666666666667*E*J_bar11*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  11] = 0.166666666666667*E*J_bar11*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  12] = 0.166666666666667*E*J_bar11*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  13] = 0.166666666666667*E*J_bar11*J_bar11 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  14] = 0.166666666666667*E*J_bar21*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  15] = 0.166666666666667*E*J_bar11*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  16] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  17] = 0.166666666666667*E*J_bar21*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  18] = 0.166666666666667*E*J_bar11*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  19] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  20] = 0.166666666666667*E*J_bar21*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  21] = 0.166666666666667*E*J_bar11*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  22] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  23] = 0.166666666666667*E*J_bar21*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  24] = 0.166666666666667*E*J_bar11*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  25] = 0.166666666666667*E*J_bar21*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  26] = 0.166666666666667*E*J_bar11*J_bar11 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  27] = 0.166666666666667*E*J_bar11*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  28] = 0.166666666666667*E*J_bar21*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  29] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  30] = 0.166666666666667*E*J_bar11*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  31] = 0.166666666666667*E*J_bar21*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  32] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  33] = 0.166666666666667*E*J_bar11*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  34] = 0.166666666666667*E*J_bar21*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  35] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  36] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  37] = 0.166666666666667*E*J_bar11*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  38] = 0.166666666666667*E*J_bar11*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  39] = 0.166666666666667*E*J_bar12*J_bar12 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  40] = 0.166666666666667*E*J_bar12*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  41] = 0.166666666666667*E*J_bar12*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  42] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  43] = 0.166666666666667*E*J_bar12*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  44] = 0.166666666666667*E*J_bar12*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  45] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  46] = 0.166666666666667*E*J_bar12*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  47] = 0.166666666666667*E*J_bar12*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  48] = 0.166666666666667*E*J_bar11*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  49] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  50] = 0.166666666666667*E*J_bar21*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  51] = 0.166666666666667*E*J_bar12*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  52] = 0.166666666666667*E*J_bar12*J_bar12 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  53] = 0.166666666666667*E*J_bar22*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  54] = 0.166666666666667*E*J_bar12*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  55] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  56] = 0.166666666666667*E*J_bar22*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  57] = 0.166666666666667*E*J_bar12*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  58] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  59] = 0.166666666666667*E*J_bar22*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  60] = 0.166666666666667*E*J_bar11*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  61] = 0.166666666666667*E*J_bar21*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  62] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  63] = 0.166666666666667*E*J_bar12*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  64] = 0.166666666666667*E*J_bar22*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  65] = 0.166666666666667*E*J_bar12*J_bar12 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  66] = 0.166666666666667*E*J_bar12*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  67] = 0.166666666666667*E*J_bar22*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  68] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  69] = 0.166666666666667*E*J_bar12*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  70] = 0.166666666666667*E*J_bar22*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  71] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  72] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  73] = 0.166666666666667*E*J_bar11*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  74] = 0.166666666666667*E*J_bar11*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  75] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  76] = 0.166666666666667*E*J_bar12*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  77] = 0.166666666666667*E*J_bar12*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  78] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  79] = 0.166666666666667*E*J_bar13*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  80] = 0.166666666666667*E*J_bar13*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  81] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  82] = 0.166666666666667*E*J_bar13*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  83] = 0.166666666666667*E*J_bar13*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  84] = 0.166666666666667*E*J_bar11*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  85] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  86] = 0.166666666666667*E*J_bar21*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  87] = 0.166666666666667*E*J_bar12*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  88] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  89] = 0.166666666666667*E*J_bar22*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  90] = 0.166666666666667*E*J_bar13*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  91] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  92] = 0.166666666666667*E*J_bar23*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  93] = 0.166666666666667*E*J_bar13*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  94] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  95] = 0.166666666666667*E*J_bar23*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  96] = 0.166666666666667*E*J_bar11*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  97] = 0.166666666666667*E*J_bar21*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  98] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  99] = 0.166666666666667*E*J_bar12*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  100] = 0.166666666666667*E*J_bar22*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  101] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  102] = 0.166666666666667*E*J_bar13*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  103] = 0.166666666666667*E*J_bar23*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  104] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  105] = 0.166666666666667*E*J_bar13*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  106] = 0.166666666666667*E*J_bar23*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  107] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  108] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  109] = 0.166666666666667*E*J_bar11*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  110] = 0.166666666666667*E*J_bar11*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  111] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  112] = 0.166666666666667*E*J_bar12*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  113] = 0.166666666666667*E*J_bar12*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  114] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  115] = 0.166666666666667*E*J_bar13*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  116] = 0.166666666666667*E*J_bar13*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  117] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  118] = 0.166666666666667*E*J_star1*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  119] = 0.166666666666667*E*J_star1*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  120] = 0.166666666666667*E*J_bar11*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  121] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  122] = 0.166666666666667*E*J_bar21*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  123] = 0.166666666666667*E*J_bar12*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  124] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  125] = 0.166666666666667*E*J_bar22*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  126] = 0.166666666666667*E*J_bar13*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  127] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  128] = 0.166666666666667*E*J_bar23*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
	in_element->local_K[  129] = 0.166666666666667*E*J_star1*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  130] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  131] = 0.166666666666667*E*J_star2*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  132] = 0.166666666666667*E*J_bar11*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  133] = 0.166666666666667*E*J_bar21*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  134] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  135] = 0.166666666666667*E*J_bar12*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  136] = 0.166666666666667*E*J_bar22*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  137] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  138] = 0.166666666666667*E*J_bar13*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  139] = 0.166666666666667*E*J_bar23*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  140] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  141] = 0.166666666666667*E*J_star1*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  142] = 0.166666666666667*E*J_star2*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	in_element->local_K[  143] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);

	//return (x14*(y24*z34 - y34*z24) - y14*(x24*z34 - z24 * x34) + z14*(x24*y34 - y24*x34));
}


//Atomic add
__device__ inline float atomicAdda(float* address, double value)

{

	float ret = atomicExch(address, 0.0f);

	float old = ret + (float)value;

	while ((old = atomicExch(address, old)) != 0.0f)

	{

		old = atomicExch(address, 0.0f) + old;

	}

	return ret;

};
__global__ void gpu_make_K(
	AFEM::element *in_vec,
	int numElem,
	int numNodes, 
	float *K_d
	)
{

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if (x < numElem){
		find_Jacobian_and_localK(&in_vec[x]);

		//K_d[x] = (in_vec[x]).local_K[0];
		int DOF[12];
		int counter = 0;
		//The two loops are responsible for finding the DOF (or q_i) for each element
		for (int npe = 0; npe < 4; npe++){
			//dummy_node = nodesInElem[nodesinelemX(npe, offset, 4)]; // The row of the matrix we looking at will be k_th element and npe (nodes per element) 	
			for (int dof = 0; dof < 3; dof++){

				DOF[counter] = in_vec[x].position_info[npe].displacement_index[dof];
				counter++;
			}
		}

		for (int c = 0; c < 12; c++){
			for (int r = 0; r < 12; r++){

				//d_A_dense[IDX2C(DOF[c], DOF[r], 3000)] = d_A_dense[IDX2C(DOF[c], DOF[r], 3000)] + E_vector[offset * 144 + c*12+r];
				atomicAdda(&(K_d[IDX2C(DOF[c], DOF[r], 3 * (numNodes))]),in_vec[x].local_K[c*12+r]);
				//IDX2C(DOF[c], DOF[r], 3000)
				//K[IDX2C(DOF[r], DOF[c], numP*dim)] = K[IDX2C(DOF[r], DOF[c], numP*dim)] + E[k][r][c];
			}
		}
		//printf("hi");
	}
	
}

//Resets the K matrix to zero
__global__ void reset_K(
	float *K_d,
	int numNodes,
	int dim

){
	int x = threadIdx.x + blockIdx.x * blockDim.x;

	if (x < numNodes*dim*numNodes*dim){
		K_d[x] = 0;
	}
}


//Allocates the cpu and gpu memory, and then copy necessary data to them
void cuda_tools::allocate_copy_CUDA_geometry_data(AFEM::element *in_array, int num_elem, int num_nodes, int dim){
	//cpu allocation of memeory

	K_h = (float*)malloc( sizeof(*K_h)*dim*num_nodes*dim*num_nodes);


	//cuda allocation of memory
	hipMalloc((void**)&elem_array_d, sizeof(AFEM::element) *num_elem); //element array
	hipMalloc((void**)&K_d, sizeof(*K_d)*dim*num_nodes*dim*num_nodes); //final global K matrix container


	//cuda copy of memory from host to device
	hipMemcpy(elem_array_d, in_array, sizeof(AFEM::element) *num_elem, hipMemcpyHostToDevice);
	hipMemset(K_d, 0, sizeof(*K_d)*dim*num_nodes*dim*num_nodes); //initialize the vector K_d to zero
	
}


void cuda_tools::copy_data_from_cuda(int num_nodes,int dim){

	
	
	
}


void cuda_tools::make_K(int num_elem,int num_nodes){
	int blocks, threads;
	if (num_elem <= 256){
		blocks = 16;
		threads = 16;
	}
	else {
		blocks = (num_elem + 1) / 256;
		threads = 256;
	}
	gpu_make_K << <blocks, threads >> > (elem_array_d, num_elem,num_nodes, K_d);
	hipMemcpy(K_h, K_d, 100, hipMemcpyDeviceToHost);

	for (int i = 0; i < 1; i++){
		std::cout << K_h[0] << " ";
	}
	//hipMemset(K_d, 0, sizeof(*K_d)*dim*num_nodes*dim*num_nodes); //initialize the vector K_d to zero

	std::cout << std::endl;
	reset_K << <blocks, threads >> >( K_d,num_nodes, 3);

}





cuda_tools::~cuda_tools(){
	free(K_h);
	hipFree(K_d);

}

__global__ void vecAdd(double *a, double *b, double *c, int n)
{
	// Get our global thread ID
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	// Make sure we do not go out of bounds
	if (id < n)
		c[id] = a[id] + b[id];
}

void hello(){
	// Size of vectors
	int n = 100000;

	// Host input vectors
	double *h_a;
	double *h_b;
	//Host output vector
	double *h_c;

	// Device input vectors
	double *d_a;
	double *d_b;
	//Device output vector
	double *d_c;

	// Size, in bytes, of each vector
	size_t bytes = n*sizeof(double);

	// Allocate memory for each vector on host
	h_a = (double*)malloc(bytes);
	h_b = (double*)malloc(bytes);
	h_c = (double*)malloc(bytes);

	// Allocate memory for each vector on GPU
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	int i;
	// Initialize vectors on host
	for (i = 0; i < n; i++) {
		h_a[i] = sin(i)*sin(i);
		h_b[i] = cos(i)*cos(i);
	}

	// Copy host vectors to device
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	int blockSize, gridSize;

	// Number of threads in each thread block
	blockSize = 1024;

	// Number of thread blocks in grid
	gridSize = (int)ceil((float)n / blockSize);

	// Execute the kernel
	vecAdd << <gridSize, blockSize >> >(d_a, d_b, d_c, n);

	// Copy array back to host
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	// Sum up vector c and print result divided by n, this should equal 1 within error
	double sum = 0;
	for (i = 0; i<n; i++)
		sum += h_c[i];
	printf("final result: %f\n", sum / n);

	// Release device memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	// Release host memory
	free(h_a);
	free(h_b);
	free(h_c);

}

//__global__ void make_K_cuda3d(double *E_vector, int *nodesInElem_device, double *x_vector, double *y_vector, double *z_vector, int *displaceInElem_device, float *d_A_dense, int *numnodes);//3D
//__global__ void make_K_cuda2d(double *K, int *nodesInElem, double *x_vector, double *y_vector, int *displaceInElem_device, float *d_A_dense, int numnodes, double thickness,double young_E,double nu,double alpha,double beta1,double beta2, double rho, double dt,double c_xi,int numE);//2D
//__global__ void make_global_K(void); 





