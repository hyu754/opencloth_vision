#include "hip/hip_runtime.h"


#include <stdio.h>


#include <math.h>

#include <iostream>
#include "AFEM_cuda.cuh"

#define nodesinelemX(node,el,nodesPerElem) (node + nodesPerElem*el) //the first entry is the element # the second entry would be the element number and the last one is the number of nodes/element
#define threeD21D(row_d,col_d,el_d,width_d,depth_d) (row_d+width_d*(col_d+depth_d*el_d)) //
#define nodesDisplacementX(dof,node,dimension) (dof + node*dimension)
#define IDX2C(i,j,ld) (((j)*(ld))+( i )) 

__global__ void hello_gpuprint(int num){
	printf("gpu print %d", &num);
}



//This is for the local K matrix
//NOTE:::: nu and E are not initilized
__device__ inline float atomicAdda(float* address, double value)

{

	float ret = atomicExch(address, 0.0f);

	float old = ret + (float)value;

	while ((old = atomicExch(address, old)) != 0.0f)

	{

		old = atomicExch(address, 0.0f) + old;

	}

	return ret;

};
__global__ void make_K_cuda3d(double *E_vector, int *nodesInElem, double *x_vector, double *y_vector, double *z_vector, int *displaceInElem_device, float *d_A_dense, int *numnodes) {
	//int x = threadIdx.x + blockIdx.x*blockDim.x; //if we have a 3D problem then this will go from 0 to 11
#if 1
	int row;
	int dummy_node;
	int loop_node;
	int dummy_row;
	int dummy_col;
	int DOF[12];
	int counter;
	int offset = threadIdx.x + blockIdx.x*blockDim.x; // offset will essentaillay be the element counter
	//int y_offset = threadIdx.y + blockIdx.y*blockDim.y;

	int max_limit = 12 * 12 * 4374;
	double E = 20000.0;
	double nu = 0.49;
	double x14 = x_vector[nodesInElem[nodesinelemX(0, offset, 4)]] - x_vector[nodesInElem[nodesinelemX(3, offset, 4)]];
	double x24 = x_vector[nodesInElem[nodesinelemX(1, offset, 4)]] - x_vector[nodesInElem[nodesinelemX(3, offset, 4)]];
	double x34 = x_vector[nodesInElem[nodesinelemX(2, offset, 4)]] - x_vector[nodesInElem[nodesinelemX(3, offset, 4)]];
	double y14 = y_vector[nodesInElem[nodesinelemX(0, offset, 4)]] - y_vector[nodesInElem[nodesinelemX(3, offset, 4)]];
	double y24 = y_vector[nodesInElem[nodesinelemX(1, offset, 4)]] - y_vector[nodesInElem[nodesinelemX(3, offset, 4)]];
	double y34 = y_vector[nodesInElem[nodesinelemX(2, offset, 4)]] - y_vector[nodesInElem[nodesinelemX(3, offset, 4)]];
	double z14 = z_vector[nodesInElem[nodesinelemX(0, offset, 4)]] - z_vector[nodesInElem[nodesinelemX(3, offset, 4)]];
	double z24 = z_vector[nodesInElem[nodesinelemX(1, offset, 4)]] - z_vector[nodesInElem[nodesinelemX(3, offset, 4)]];
	double z34 = z_vector[nodesInElem[nodesinelemX(2, offset, 4)]] - z_vector[nodesInElem[nodesinelemX(3, offset, 4)]];

	//std::cout << x14*(y24*z34 - y34*z24) - y14*(x24*z34 - z24 * 34) + z14*(x24*y34 - y24*x34) << std::endl;
	double det_J = (x14*(y24*z34 - y34*z24) - y14*(x24*z34 - z24 * x34) + z14*(x24*y34 - y24*x34));

	double J_bar11 = (y24*z34 - z24*y34) / det_J;
	double J_bar12 = (z14*y34 - y14*z34) / det_J;
	double J_bar13 = (y14*z24 - z14*y24) / det_J;
	double J_bar21 = (z24*x34 - x24*z34) / det_J;
	double J_bar22 = (x14*z34 - z14*x34) / det_J;
	double J_bar23 = (z14*x24 - x14*z24) / det_J;
	double J_bar31 = (x24*y34 - y24*x34) / det_J;
	double J_bar32 = (y14*x34 - x14*y34) / det_J;
	double J_bar33 = (x14*y24 - y14*x24) / det_J;

	double J_star1 = -(J_bar11 + J_bar12 + J_bar13);
	double J_star2 = -(J_bar21 + J_bar22 + J_bar23);
	double J_star3 = -(J_bar31 + J_bar32 + J_bar33);



	E_vector[offset * 144 + 0] = 0.166666666666667*E*J_bar11*J_bar11 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 1] = 0.166666666666667*E*J_bar11*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 2] = 0.166666666666667*E*J_bar11*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 3] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 4] = 0.166666666666667*E*J_bar11*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 5] = 0.166666666666667*E*J_bar11*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 6] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 7] = 0.166666666666667*E*J_bar11*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 8] = 0.166666666666667*E*J_bar11*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 9] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 10] = 0.166666666666667*E*J_bar11*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 11] = 0.166666666666667*E*J_bar11*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 12] = 0.166666666666667*E*J_bar11*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 13] = 0.166666666666667*E*J_bar11*J_bar11 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 14] = 0.166666666666667*E*J_bar21*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 15] = 0.166666666666667*E*J_bar11*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 16] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 17] = 0.166666666666667*E*J_bar21*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 18] = 0.166666666666667*E*J_bar11*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 19] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 20] = 0.166666666666667*E*J_bar21*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 21] = 0.166666666666667*E*J_bar11*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 22] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 23] = 0.166666666666667*E*J_bar21*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 24] = 0.166666666666667*E*J_bar11*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar11*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 25] = 0.166666666666667*E*J_bar21*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 26] = 0.166666666666667*E*J_bar11*J_bar11 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar21 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar31 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 27] = 0.166666666666667*E*J_bar11*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 28] = 0.166666666666667*E*J_bar21*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 29] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 30] = 0.166666666666667*E*J_bar11*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 31] = 0.166666666666667*E*J_bar21*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 32] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 33] = 0.166666666666667*E*J_bar11*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 34] = 0.166666666666667*E*J_bar21*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 35] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 36] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 37] = 0.166666666666667*E*J_bar11*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 38] = 0.166666666666667*E*J_bar11*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 39] = 0.166666666666667*E*J_bar12*J_bar12 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 40] = 0.166666666666667*E*J_bar12*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 41] = 0.166666666666667*E*J_bar12*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 42] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 43] = 0.166666666666667*E*J_bar12*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 44] = 0.166666666666667*E*J_bar12*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 45] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 46] = 0.166666666666667*E*J_bar12*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 47] = 0.166666666666667*E*J_bar12*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 48] = 0.166666666666667*E*J_bar11*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 49] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 50] = 0.166666666666667*E*J_bar21*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 51] = 0.166666666666667*E*J_bar12*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 52] = 0.166666666666667*E*J_bar12*J_bar12 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 53] = 0.166666666666667*E*J_bar22*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 54] = 0.166666666666667*E*J_bar12*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 55] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 56] = 0.166666666666667*E*J_bar22*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 57] = 0.166666666666667*E*J_bar12*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 58] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 59] = 0.166666666666667*E*J_bar22*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 60] = 0.166666666666667*E*J_bar11*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 61] = 0.166666666666667*E*J_bar21*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 62] = 0.166666666666667*E*J_bar11*J_bar12*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar32*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 63] = 0.166666666666667*E*J_bar12*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar12*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 64] = 0.166666666666667*E*J_bar22*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 65] = 0.166666666666667*E*J_bar12*J_bar12 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar22 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar32 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 66] = 0.166666666666667*E*J_bar12*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 67] = 0.166666666666667*E*J_bar22*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 68] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 69] = 0.166666666666667*E*J_bar12*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 70] = 0.166666666666667*E*J_bar22*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 71] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 72] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 73] = 0.166666666666667*E*J_bar11*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 74] = 0.166666666666667*E*J_bar11*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 75] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 76] = 0.166666666666667*E*J_bar12*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 77] = 0.166666666666667*E*J_bar12*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 78] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 79] = 0.166666666666667*E*J_bar13*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 80] = 0.166666666666667*E*J_bar13*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 81] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 82] = 0.166666666666667*E*J_bar13*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 83] = 0.166666666666667*E*J_bar13*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 84] = 0.166666666666667*E*J_bar11*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar21*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 85] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 86] = 0.166666666666667*E*J_bar21*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 87] = 0.166666666666667*E*J_bar12*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar22*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 88] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 89] = 0.166666666666667*E*J_bar22*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 90] = 0.166666666666667*E*J_bar13*J_bar23*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 91] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 92] = 0.166666666666667*E*J_bar23*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 93] = 0.166666666666667*E*J_bar13*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 94] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 95] = 0.166666666666667*E*J_bar23*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 96] = 0.166666666666667*E*J_bar11*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 97] = 0.166666666666667*E*J_bar21*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar31*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 98] = 0.166666666666667*E*J_bar11*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 99] = 0.166666666666667*E*J_bar12*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 100] = 0.166666666666667*E*J_bar22*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar32*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 101] = 0.166666666666667*E*J_bar12*J_bar13*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_bar23*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_bar33*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 102] = 0.166666666666667*E*J_bar13*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar13*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 103] = 0.166666666666667*E*J_bar23*J_bar33*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar33*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 104] = 0.166666666666667*E*J_bar13*J_bar13 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_bar23 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_bar33 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 105] = 0.166666666666667*E*J_bar13*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 106] = 0.166666666666667*E*J_bar23*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 107] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 108] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 109] = 0.166666666666667*E*J_bar11*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 110] = 0.166666666666667*E*J_bar11*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 111] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 112] = 0.166666666666667*E*J_bar12*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 113] = 0.166666666666667*E*J_bar12*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 114] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 115] = 0.166666666666667*E*J_bar13*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 116] = 0.166666666666667*E*J_bar13*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 117] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 118] = 0.166666666666667*E*J_star1*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 119] = 0.166666666666667*E*J_star1*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 120] = 0.166666666666667*E*J_bar11*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 121] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 122] = 0.166666666666667*E*J_bar21*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 123] = 0.166666666666667*E*J_bar12*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 124] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 125] = 0.166666666666667*E*J_bar22*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 126] = 0.166666666666667*E*J_bar13*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 127] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 128] = 0.166666666666667*E*J_bar23*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*nu / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 129] = 0.166666666666667*E*J_star1*J_star2*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 130] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 131] = 0.166666666666667*E*J_star2*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 132] = 0.166666666666667*E*J_bar11*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 133] = 0.166666666666667*E*J_bar21*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 134] = 0.166666666666667*E*J_bar11*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar21*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar31*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 135] = 0.166666666666667*E*J_bar12*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 136] = 0.166666666666667*E*J_bar22*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 137] = 0.166666666666667*E*J_bar12*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar22*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar32*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 138] = 0.166666666666667*E*J_bar13*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 139] = 0.166666666666667*E*J_bar23*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 140] = 0.166666666666667*E*J_bar13*J_star1*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar23*J_star2*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_bar33*J_star3*det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 141] = 0.166666666666667*E*J_star1*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star1*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 142] = 0.166666666666667*E*J_star2*J_star3*det_J*nu / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star3*det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1);
	E_vector[offset * 144 + 143] = 0.166666666666667*E*J_star1*J_star1 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star2*J_star2 * det_J*(-0.5*nu + 0.5) / (-2 * nu*nu - nu + 1) + 0.166666666666667*E*J_star3*J_star3 * det_J*(-nu + 1.0) / (-2 * nu*nu - nu + 1);


	counter = 0;
	//The two loops are responsible for finding the DOF (or q_i) for each element
	for (int npe = 0; npe < 4; npe++){
		dummy_node = nodesInElem[nodesinelemX(npe, offset, 4)]; // The row of the matrix we looking at will be k_th element and npe (nodes per element) 	
		for (int dof = 0; dof < 3; dof++){

			DOF[counter] = displaceInElem_device[nodesDisplacementX(dof, dummy_node, 3)];
			counter++;
		}
	}

	//we will use atomic add because we will be writting to a single location multiple times (perhaps) 
	for (int c = 0; c < 12; c++){
		for (int r = 0; r < 12; r++){

			//d_A_dense[IDX2C(DOF[c], DOF[r], 3000)] = d_A_dense[IDX2C(DOF[c], DOF[r], 3000)] + E_vector[offset * 144 + c*12+r];
			atomicAdda(&(d_A_dense[IDX2C(DOF[c], DOF[r], 3 * (*numnodes))]), E_vector[offset * 144 + c * 12 + r]);
			//IDX2C(DOF[c], DOF[r], 3000)
			//K[IDX2C(DOF[r], DOF[c], numP*dim)] = K[IDX2C(DOF[r], DOF[c], numP*dim)] + E[k][r][c];
		}
	}

#endif // 0




}

__global__ void vecAdd(double *a, double *b, double *c, int n)
{
	// Get our global thread ID
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	// Make sure we do not go out of bounds
	if (id < n)
		c[id] = a[id] + b[id];
}

void cuda_tools::hello(){
	// Size of vectors
	int n = 100000;

	// Host input vectors
	double *h_a;
	double *h_b;
	//Host output vector
	double *h_c;

	// Device input vectors
	double *d_a;
	double *d_b;
	//Device output vector
	double *d_c;

	// Size, in bytes, of each vector
	size_t bytes = n*sizeof(double);

	// Allocate memory for each vector on host
	h_a = (double*)malloc(bytes);
	h_b = (double*)malloc(bytes);
	h_c = (double*)malloc(bytes);

	// Allocate memory for each vector on GPU
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	int i;
	// Initialize vectors on host
	for (i = 0; i < n; i++) {
		h_a[i] = sin(i)*sin(i);
		h_b[i] = cos(i)*cos(i);
	}

	// Copy host vectors to device
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	int blockSize, gridSize;

	// Number of threads in each thread block
	blockSize = 1024;

	// Number of thread blocks in grid
	gridSize = (int)ceil((float)n / blockSize);

	// Execute the kernel
	vecAdd << <gridSize, blockSize >> >(d_a, d_b, d_c, n);

	// Copy array back to host
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	// Sum up vector c and print result divided by n, this should equal 1 within error
	double sum = 0;
	for (i = 0; i<n; i++)
		sum += h_c[i];
	printf("final result: %f\n", sum / n);

	// Release device memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	// Release host memory
	free(h_a);
	free(h_b);
	free(h_c);

}

//__global__ void make_K_cuda3d(double *E_vector, int *nodesInElem_device, double *x_vector, double *y_vector, double *z_vector, int *displaceInElem_device, float *d_A_dense, int *numnodes);//3D
//__global__ void make_K_cuda2d(double *K, int *nodesInElem, double *x_vector, double *y_vector, int *displaceInElem_device, float *d_A_dense, int numnodes, double thickness,double young_E,double nu,double alpha,double beta1,double beta2, double rho, double dt,double c_xi,int numE);//2D
//__global__ void make_global_K(void); 





